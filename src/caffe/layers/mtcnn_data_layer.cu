#ifdef USE_OPENCV
#include <opencv2/core/core.hpp>
#endif  // USE_OPENCV
#include <stdint.h>

#include <boost/thread.hpp>
#include <vector>

#include "caffe/data_transformer.hpp"
#include "caffe/layers/mtcnn_data_layer.hpp"
#include "caffe/util/benchmark.hpp"

#include "caffe/blob.hpp"
#include "caffe/data_transformer.hpp"
#include "caffe/internal_thread.hpp"
#include "caffe/layer.hpp"
#include "caffe/layers/base_data_layer.hpp"
#include "caffe/proto/caffe.pb.h"
#include "caffe/util/blocking_queue.hpp"

namespace caffe {

template <typename Dtype>
void MTCNNDataLayer<Dtype>::Forward_gpu(
    const vector<Blob<Dtype>*>& bottom, const vector<Blob<Dtype>*>& top) {

  Batch<Dtype>* batch = prefetch_full_.pop("Data layer prefetch queue empty");
  // Reshape to loaded data.
  top[0]->ReshapeLike(batch->data_);
  // Copy the data
  caffe_copy(batch->data_.count(), batch->data_.cpu_data(),
	  top[0]->mutable_cpu_data());
  DLOG(INFO) << "Prefetch copied";

  //label
  top[1]->ReshapeLike(batch->label_);
  caffe_copy(batch->label_.count(), batch->label_.cpu_data(),
	  top[1]->mutable_cpu_data());

  //roi
  top[2]->ReshapeLike(batch->roi_);
  caffe_copy(batch->roi_.count(), batch->roi_.cpu_data(),
	  top[2]->mutable_cpu_data());

  if (output_pts_){
	  //pts
	  top[3]->ReshapeLike(batch->pts_);
	  caffe_copy(batch->pts_.count(), batch->pts_.cpu_data(),
		  top[3]->mutable_cpu_data());
  }

  CUDA_CHECK(hipStreamSynchronize(hipStreamDefault));
  prefetch_free_.push(batch);
}

template <typename Dtype>
void MTCNNDataLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
	const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom){}
INSTANTIATE_LAYER_GPU_FUNCS(MTCNNDataLayer);

}  // namespace caffe
