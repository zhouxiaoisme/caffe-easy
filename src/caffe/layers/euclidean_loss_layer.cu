#include <vector>

#include "caffe/layers/euclidean_loss_layer.hpp"
#include "caffe/util/math_functions.hpp"

namespace caffe {

template <typename Dtype>
void EuclideanLossLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {
  
#if 0
	int count = bottom[0]->count();
  caffe_gpu_sub(
      count,
      bottom[0]->gpu_data(),
      bottom[1]->gpu_data(),
      diff_.mutable_gpu_data());
  Dtype dot;
  caffe_gpu_dot(count, diff_.gpu_data(), diff_.gpu_data(), &dot);
  Dtype loss = dot / bottom[0]->num() / Dtype(2);
  top[0]->mutable_cpu_data()[0] = loss;
#endif



  if (bottom.size() == 2){
	  int count = bottom[0]->count();
	  caffe_gpu_sub(
		  count,
		  bottom[0]->gpu_data(),
		  bottom[1]->gpu_data(),
		  diff_.mutable_gpu_data());
	  Dtype dot;
	  caffe_gpu_dot(count, diff_.gpu_data(), diff_.gpu_data(), &dot);
	  Dtype loss = dot / bottom[0]->num() / Dtype(2);
	  top[0]->mutable_cpu_data()[0] = loss;
	  num_labels = bottom[0]->num();
  }
  else if (bottom.size() == 3){
	  const Dtype* a = bottom[0]->gpu_data();
	  const Dtype* b = bottom[1]->gpu_data();
	  const Dtype* label = bottom[2]->cpu_data();
	  Dtype* diff = diff_.mutable_gpu_data();
	  int channels = bottom[0]->channels();
	  int num = bottom[0]->num();
	  int w = bottom[0]->width();
	  int h = bottom[0]->height();
	  int plane = w * h;
	  Dtype dot = 0;
	  num_labels = 0;

	  //printf("num = %d, w = %d, h = %d\n", num, w, h);
	  caffe_gpu_memset(sizeof(Dtype)*bottom[0]->count(), 0, diff);

	  //ͨ����ͬ
	  for (int n = 0; n < num; ++n){
		  for (int i = 0; i < w; ++i){
			  for (int j = 0; j < h; ++j){
				  Dtype v = *(label + i + j * w + n * w * h);
				  if (v != 0){
					  for (int c = 0; c < channels; ++c){
						  num_labels++;
						  const Dtype* pa = a + i + j * w + n * channels * w * h + w * h * c;
						  const Dtype* pb = b + i + j * w + n * channels * w * h + w * h * c;
						  Dtype* pdiff = diff + i + j * w + n * channels * w * h + w * h * c;
						  //*pdiff = (*pa - *pb) * (*pa - *pb);
						  //dot += *pdiff;

						  caffe_gpu_sub(1, pa, pb, pdiff);
					  }
				  }
			  }
		  }
	  }

	  caffe_gpu_dot(bottom[0]->count(), diff, diff, &dot);
	 // printf("num_labels = %d, dot = %f\n", num_labels, dot);
	  Dtype loss = num_labels == 0 ? 0 : dot / num_labels / Dtype(2);
	  top[0]->mutable_cpu_data()[0] = loss;
  }
}

template <typename Dtype>
void EuclideanLossLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
  for (int i = 0; i < 2; ++i) {
    if (propagate_down[i]) {

	  //printf("num_labels2 = %d\n", num_labels);
      const Dtype sign = (i == 0) ? 1 : -1;
	  const Dtype alpha = sign * top[0]->cpu_diff()[0] / num_labels;
      caffe_gpu_axpby(
          bottom[i]->count(),              // count
          alpha,                              // alpha
          diff_.gpu_data(),                   // a
          Dtype(0),                           // beta
          bottom[i]->mutable_gpu_diff());  // b
    }
  }
}

INSTANTIATE_LAYER_GPU_FUNCS(EuclideanLossLayer);

}  // namespace caffe
